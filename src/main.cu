#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include ""
#include "Compact.cuh"

__global__ void prediction(char* data, int* pred) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (data[id] > 'f' &&
        data[id] < 'l')
        pred[id] = 1;
    else pred[id] = 0;
}

int main()
{
    hipSetDevice(0);
    const char* abc = "abcdefghijklmnopqrstuvwxyz";

    printf("%s\n", Compact::compress(abc, strlen(abc), prediction));

    hipDeviceReset();

    return 0;
}
