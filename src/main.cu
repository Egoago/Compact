#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include ""
#include "Compact.cuh"

__global__ void prediction(char* data, int* pred) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    pred[id] = (data[id] >= 'f' &&
                data[id] <= 'l');
}

int main()
{
    hipSetDevice(0);
    const char* abc = "abcdefghijklmnopqrstuvwxyz";

    printf("%s\n", Compact::compress(abc, (unsigned int)strlen(abc), prediction));

    hipDeviceReset();

    return 0;
}
