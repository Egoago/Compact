#include "hip/hip_runtime.h"
﻿#include "compact.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void sum(int* pred, int elementCount, int* sum) {
    int id = threadIdx.x;
    if(id < elementCount)
        sum[id] = pred[id];
    __syncthreads();
    if (id < elementCount)
        for (int i = 1; i < elementCount; i *= 2) {
            int tmp = sum[id];
            if (id + i < elementCount)
                sum[id + i] += tmp;
            __syncthreads();
        }
}

__global__ void gather(void* data, int* pred, int* offset, void* result, unsigned int elementSize, unsigned int elementCount) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (elementCount > id && pred[id] == 1)
        memcpy((char*)result + (offset[id] - 1) * elementSize, (char*)data + (id * elementSize), elementSize);
}

unsigned int Compact::operator()(void** compressedData)
{
    hipError_t cudaStatus;

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "beginning: %s\n", hipGetErrorString(cudaStatus));
    }
    cudaStatus = hipMalloc(&dataGPU, elementSize * elementCount);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMalloc1 failed!\n");
        goto Error;
    }
    cudaStatus = hipMalloc(&predGPU, sizeof(int) * elementCount);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMalloc2 failed!\n");
        goto Error;
    }
    cudaStatus = hipMalloc(&offsetGPU, sizeof(int) * elementCount);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMalloc3 failed!\n");
        goto Error;
    }

    cudaStatus = hipMemcpy(dataGPU, dataCPU, elementSize * elementCount, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy1 failed!\n");
        goto Error;
    }

    predictor <<<(elementCount + 63) / 64, 64 >>> (dataGPU, predGPU, elementCount);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "predictor launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    sum << <1, elementCount>> > (predGPU, elementCount, offsetGPU);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "sum launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    hipMemcpy(&predCount, &offsetGPU[elementCount - 1], sizeof(int), hipMemcpyDeviceToHost);

    cudaStatus = hipMalloc(&resultGPU, elementSize * predCount);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMalloc4 failed!\n");
        goto Error;
    }

    gather << <(elementCount + 63) / 64, 64 >> > (dataGPU, predGPU, offsetGPU, resultGPU, elementSize, elementCount);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "gather launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    *compressedData = new char[elementSize * predCount];
    cudaStatus = hipMemcpy(*compressedData, resultGPU, elementSize * predCount, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "memcpy failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

Error:
    hipFree(dataGPU);
    hipFree(predGPU);
    hipFree(offsetGPU);
    hipFree(resultGPU);
    dataGPU = resultGPU = nullptr;
    predGPU = offsetGPU = nullptr;
    return predCount;
}