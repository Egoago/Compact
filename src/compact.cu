#include "hip/hip_runtime.h"
﻿#include "compact.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void sum(int* pred, int elementCount, int* sum) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    sum[id] = pred[id];
    __syncthreads();
    for (int i = 1; i < elementCount; i *= 2) {
        int tmp = sum[id];
        if (id + i < elementCount)
            sum[id + i] += tmp;
        __syncthreads();
    }
}

__global__ void gather(void* data, int* pred, int* offset, void* result, unsigned int elementSize) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (pred[id] == 1)
        memcpy((char*)result + (offset[id] - 1) * elementSize, (char*)data + (id * elementSize), elementSize);
}

size_t Compact::operator()(void** compressedData)
{
    hipMalloc(&dataGPU, elementSize * elementCount);
    hipMalloc(&predGPU, sizeof(int) * elementCount);
    hipMalloc(&offsetGPU, sizeof(int) * elementCount);
    hipMemcpy(dataGPU, dataCPU, elementSize * elementCount, hipMemcpyHostToDevice);

    predictor << <1, elementCount >> > (dataGPU, predGPU);

    sum << <1, elementCount >> > (predGPU, elementCount, offsetGPU);

    hipMemcpy(&predCount, &offsetGPU[elementCount - 1], sizeof(int), hipMemcpyDeviceToHost);
    hipMalloc(&resultGPU, elementSize * predCount);

    gather << <1, elementCount >> > (dataGPU, predGPU, offsetGPU, resultGPU, elementSize);

    *compressedData = new char[elementSize * predCount];
    hipMemcpy(*compressedData, resultGPU, elementSize * predCount, hipMemcpyDeviceToHost);

    hipFree(dataGPU);
    hipFree(predGPU);
    hipFree(offsetGPU);
    hipFree(resultGPU);
    //hipDeviceSynchronize();
    return predCount;
}