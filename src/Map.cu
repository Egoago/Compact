#include "hip/hip_runtime.h"
﻿#include "Map.cuh"

__global__ void square(int* dataGPU, int dataSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    dataGPU[index] = dataGPU[index] * dataGPU[index];
}

void map(int* b, const int* a, unsigned int size) {
    int* dataGPU;
    hipMalloc(&dataGPU, sizeof(int) * size);
    hipMemcpy(dataGPU, a, sizeof(int) * size, hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = 4;
    square <<<blocksPerGrid, threadsPerBlock >>> (dataGPU, size);
    hipMemcpy(b, dataGPU, sizeof(int) * size, hipMemcpyDeviceToHost);
    hipFree(dataGPU);
}