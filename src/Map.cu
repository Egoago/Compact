#include "hip/hip_runtime.h"
﻿#include "Map.cuh"
#include ""
#include "hip/hip_runtime.h"

__global__ void square(int* dataGPU) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    dataGPU[index] = dataGPU[index] * dataGPU[index];
}

void map(int* b, const int* a, unsigned int size) {
    int* dataGPU;
    hipMalloc(&dataGPU, sizeof(int) * size);
    hipMemcpy(dataGPU, a, sizeof(int) * size, hipMemcpyHostToDevice);
    int threadsPerBlock = size;
    square <<<1, size >>> (dataGPU);
    hipMemcpy(b, dataGPU, sizeof(int) * size, hipMemcpyDeviceToHost);
    hipFree(dataGPU);
}